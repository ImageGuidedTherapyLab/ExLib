        #include <thrust/version.h>
        #include <cusp/version.h>
        #include <iostream>
        
        int main(void)
        {
            int thrust_major = THRUST_MAJOR_VERSION;
            int thrust_minor = THRUST_MINOR_VERSION;
        
            int cusp_major = CUSP_MAJOR_VERSION;
            int cusp_minor = CUSP_MINOR_VERSION;
            int cusp_subminor = CUSP_SUBMINOR_VERSION;

            int ierr=hipDeviceReset();
            std::cout << "Thrust v" << thrust_major << "." << thrust_minor << std::endl;
            std::cout << "Cusp   v" << cusp_major << "." << cusp_minor << "." << cusp_subminor << std::endl;
            int numdev = 0;
            ierr=hipGetDeviceCount(&numdev);
            std::cout << "ierr" << ierr  << std::endl;
            std::cout << "dev " << numdev<< std::endl;
        
            return 0;
        }

