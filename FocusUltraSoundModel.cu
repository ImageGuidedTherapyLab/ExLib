#include "hip/hip_runtime.h"
static char help[] = "Solves -Laplacian u - exp(u) = 0,  0 < x < 1 using GPU\n\n";
/*
   Same as ex47.c except it also uses the GPU to evaluate the function
*/

#include <petscdmda.h>
#include <petscsnes.h>
#include <petsccusp.h>
#include "cusp/detail/device/utils.h"

extern PetscErrorCode ComputeFunction(SNES,Vec,Vec,void*), ComputeJacobian(SNES,Vec,Mat*,Mat*,MatStructure*,void*);
PetscBool  useCUSP = PETSC_FALSE;
PetscLogEvent LogFunction = 0;
__device__ PetscInt *cudaTest;

struct StarStencil
{
  PetscInt       m_rank,m_deviceNum; //device info
  PetscInt       m_xs,m_ys,m_zs,m_xm,m_ym,m_zm; //corners
  PetscScalar    m_hx,m_hy,m_hz;
  PetscScalar    m_x0,m_y0,m_z0;
  PetscScalar    m_density           ;
  PetscScalar    m_specificheat      ;
  PetscScalar    m_deltat            ;
  PetscScalar    m_conduction        ;
  PetscScalar    m_bloodspecificheat ;
  PetscScalar    m_perfusion         ;
  PetscScalar    m_bodytemp          ;
  
  StarStencil(PetscInt rank, PetscInt deviceNum,
              PetscInt  xs,PetscInt  ys,PetscInt  zs, 
              PetscInt  xm,PetscInt  ym,PetscInt  zm,
              PetscScalar hx,PetscScalar hy,PetscScalar hz) : 
               m_rank(rank),m_deviceNum(deviceNum),
               m_xs(xs),m_ys(ys),m_zs(zs), 
               m_xm(xm),m_ym(ym),m_zm(zm), 
               m_hx(hx),m_hy(hy),m_hz(hz) 
               {
                  m_density           = 1.e3;
                  m_specificheat      = 3.8e3;
                  m_deltat            = 1.00;
                  m_conduction        = 0.57;
                  m_bloodspecificheat = 3.4e3;
                  m_perfusion         =  6.0;
                  m_bodytemp          = 37.0;
                  m_x0          = 0.005;
                  m_y0          = 0.005;
                  m_z0          = 0.005;
               }

	template <typename Tuple>
	__host__ __device__
	void operator()(Tuple t)
	{
		/* f = (2*u_i - u_(i+1) - u_(i-1))/h - h*exp(u_i) */
	     thrust::get<0>(t) = 1;
             PetscInt Iz = thrust::get<8>(t)/m_ym/m_xm;
             PetscInt Iy = (thrust::get<8>(t)-Iz*m_ym*m_xm)/m_xm;
             PetscInt Ix = (thrust::get<8>(t)-Iz*m_ym*m_xm- Iy*m_xm);
             PetscScalar sc      = m_hx*m_hz*m_hy;
             PetscScalar hxhzdhy = m_hx*m_hz/m_hy;
             PetscScalar hyhzdhx = m_hy*m_hz/m_hx;
             PetscScalar hxhydhz = m_hx*m_hy/m_hz;
             PetscScalar two     = 2.0;
             PetscScalar u_val       = thrust::get<1>(t) ;//1  u(i  ,j  ,k  )
             // print launch parameters and dbg info
             // printf("rank=%d device=%d blockDim=(%d,%d,%d) gridDim=(%d,%d,%d) warpSize=%d blockIdx=(%d,%d,%d) threadIdx=(%d,%d,%d) size=(%d,%d,%d) globalID=%d index=(%d,%d,%d)\n",m_rank,m_deviceNum,blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z, warpSize,blockIdx.x,blockIdx.y,blockIdx.z,threadIdx.x,threadIdx.y,threadIdx.z,m_xm,m_ym,m_zm,thrust::get<8>(t),Ix,Iy,Iz);
             if (
                 Ix > 0  && Ix < m_xm-1
                         &&
                 Iy > 0  && Iy < m_ym-1
                         &&
                 Iz > 0  && Iz < m_zm-1
                ) {
               // decode the tuple
               PetscScalar u_east      = thrust::get<2>(t) ;//2  u(i+1,j  ,k  )
               PetscScalar u_west      = thrust::get<3>(t) ;//3  u(i-1,j  ,k  )
               PetscScalar u_north     = thrust::get<4>(t) ;//4  u(i  ,j+1,k  )
               PetscScalar u_south     = thrust::get<5>(t) ;//5  u(i  ,j-1,k  )
               PetscScalar u_up        = thrust::get<6>(t) ;//6  u(i  ,j  ,k+1)
               PetscScalar u_down      = thrust::get<7>(t) ;//7  u(i  ,j  ,k-1)
               PetscScalar u_xx        = (-u_east  + two*u_val - u_west )*hyhzdhx;
               PetscScalar u_yy        = (-u_north + two*u_val - u_south)*hxhzdhy;
               PetscScalar u_zz        = (-u_up    + two*u_val - u_down )*hxhydhz;
               PetscScalar sqdist      = (m_hx * Ix - m_x0)*(m_hx * Ix - m_x0)
                                       + (m_hy * Iy - m_y0)*(m_hy * Iy - m_y0)
                                       + (m_hz * Iz - m_z0)*(m_hz * Iz - m_z0);
               PetscScalar source      = 1.e4 * exp(5.0/(sqdist +1.0));
               thrust::get<0>(t) = sc * ( source
                              + m_density*m_specificheat/m_deltat* u_val 
                              + m_bloodspecificheat*m_perfusion*(m_bodytemp - 0.5*u_val) ) 
                              + m_conduction/2.0* (u_xx + u_yy + u_zz) ;
             } else { // dirichlet bc everywhere else
               thrust::get<0>(t) = u_val;
             } 
		
	}
};
int main(int argc,char **argv) 
{
  SNES           snes; 
  Vec            x,f;  
  Mat            J;
  PetscErrorCode ierr;
  hipError_t      ierrCuda;
  char           *tmp,typeName[256];
  int            myrank;
  PetscBool      flg;

  PetscInitialize(&argc,&argv,(char *)0,help);

  MPI_Comm_rank(PETSC_COMM_WORLD, &myrank);
  int deviceNum=myrank;
  {
    int deviceCount;
    CUDA_SAFE_CALL(hipGetDeviceCount(&deviceCount));
    
    ierr = PetscPrintf(PETSC_COMM_SELF, "!!!!!found %d devices !!!!!\n",deviceCount);CHKERRQ(ierr);
    if (deviceCount == 0) {
      ierr = PetscPrintf(PETSC_COMM_SELF, "!!!!!No devices found!!!!!\n");CHKERRQ(ierr);
      return -1000;
    }

    if (deviceNum >= deviceCount || deviceNum < 0) {
      ierr = PetscPrintf(PETSC_COMM_SELF, "\n!!!!!Invalid GPU number %d given hence default gpu %d will be used !!!!!\n", deviceNum, 0);CHKERRQ(ierr);
      deviceNum = 0;
    }
  }

  ierrCuda =  hipSetDevice(deviceNum);
  if (ierrCuda != hipSuccess) {
    ierr = PetscPrintf(PETSC_COMM_SELF, " cuda Error: %s , exiting\n",hipGetErrorString( ierrCuda));CHKERRQ(ierr);
    return -1;
  }
  ierr = PetscPrintf(PETSC_COMM_SELF, " reseting GPU: \n");CHKERRQ(ierr);
  CUDA_SAFE_CALL(hipDeviceReset());

  ierr = PetscPrintf(PETSC_COMM_SELF, "Running on...\n\n");CHKERRQ(ierr);
  hipDeviceProp_t deviceProp;
  if (hipGetDeviceProperties(&deviceProp, deviceNum) == hipSuccess) {
    ierr = PetscPrintf(PETSC_COMM_SELF, " Device %d: %s %d.%d\n", deviceNum, deviceProp.name,deviceProp.major,deviceProp.minor);CHKERRQ(ierr);
    ierr = PetscPrintf(PETSC_COMM_SELF," Global memory available on device in bytes %d\n"                            ,  deviceProp.totalGlobalMem                  );
    ierr = PetscPrintf(PETSC_COMM_SELF," Shared memory available per block in bytes %d\n"                            ,  deviceProp.sharedMemPerBlock               );
    ierr = PetscPrintf(PETSC_COMM_SELF," 32-bit registers available per block %d\n"                                  ,  deviceProp.regsPerBlock                    );
    ierr = PetscPrintf(PETSC_COMM_SELF," Warp size in threads %d\n"                                                  ,  deviceProp.warpSize                        );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum pitch in bytes allowed by memory copies %d\n"                       ,  deviceProp.memPitch                        );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum number of threads per block %d\n"                                   ,  deviceProp.maxThreadsPerBlock              );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum size of each dimension of a block %d\n"                             ,  deviceProp.maxThreadsDim[0]                );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum size of each dimension of a block %d\n"                             ,  deviceProp.maxThreadsDim[1]                );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum size of each dimension of a block %d\n"                             ,  deviceProp.maxThreadsDim[2]                );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum size of each dimension of a grid %d\n"                              ,  deviceProp.maxGridSize[0]                  );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum size of each dimension of a grid %d\n"                              ,  deviceProp.maxGridSize[1]                  );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum size of each dimension of a grid %d\n"                              ,  deviceProp.maxGridSize[2]                  );
    ierr = PetscPrintf(PETSC_COMM_SELF," Clock frequency in kilohertz %d\n"                                          ,  deviceProp.clockRate                       );
    ierr = PetscPrintf(PETSC_COMM_SELF," Constant memory available on device in bytes %d\n"                          ,  deviceProp.totalConstMem                   );
    ierr = PetscPrintf(PETSC_COMM_SELF," Alignment requirement for textures %d\n"                                    ,  deviceProp.textureAlignment                );
    ierr = PetscPrintf(PETSC_COMM_SELF," Number of multiprocessors on device %d\n"                                   ,  deviceProp.multiProcessorCount             );
    ierr = PetscPrintf(PETSC_COMM_SELF," Specified whether there is a run time limit on kernels %d\n"                ,  deviceProp.kernelExecTimeoutEnabled        );
    ierr = PetscPrintf(PETSC_COMM_SELF," Device is integrated as opposed to discrete %d\n"                           ,  deviceProp.integrated                      );
    ierr = PetscPrintf(PETSC_COMM_SELF," Device can map host memory with hipHostAlloc/hipHostGetDevicePointer %d\n",  deviceProp.canMapHostMemory                );
    ierr = PetscPrintf(PETSC_COMM_SELF," Compute mode (See ::hipComputeMode) %d\n"                                  ,  deviceProp.computeMode                     );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 1D texture size %d\n"                                               ,  deviceProp.maxTexture1D                    );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 2D texture dimensions %d\n"                                         ,  deviceProp.maxTexture2D[0]                 );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 2D texture dimensions %d\n"                                         ,  deviceProp.maxTexture2D[1]                 );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 3D texture dimensions %d\n"                                         ,  deviceProp.maxTexture3D[0]                 );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 3D texture dimensions %d\n"                                         ,  deviceProp.maxTexture3D[1]                 );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 3D texture dimensions %d\n"                                         ,  deviceProp.maxTexture3D[2]                 );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 1D layered texture dimensions %d\n"                                 ,  deviceProp.maxTexture1DLayered[0]          );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 1D layered texture dimensions %d\n"                                 ,  deviceProp.maxTexture1DLayered[1]          );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 2D layered texture dimensions %d\n"                                 ,  deviceProp.maxTexture2DLayered[0]          );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 2D layered texture dimensions %d\n"                                 ,  deviceProp.maxTexture2DLayered[1]          );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 2D layered texture dimensions %d\n"                                 ,  deviceProp.maxTexture2DLayered[2]          );
    ierr = PetscPrintf(PETSC_COMM_SELF," Alignment requirements for surfaces %d\n"                                   ,  deviceProp.surfaceAlignment                );
    ierr = PetscPrintf(PETSC_COMM_SELF," Device can possibly execute multiple kernels concurrently %d\n"             ,  deviceProp.concurrentKernels               );
    ierr = PetscPrintf(PETSC_COMM_SELF," Device has ECC support enabled %d\n"                                        ,  deviceProp.ECCEnabled                      );
    ierr = PetscPrintf(PETSC_COMM_SELF," PCI bus ID of the device %d\n"                                              ,  deviceProp.pciBusID                        );
    ierr = PetscPrintf(PETSC_COMM_SELF," PCI device ID of the device %d\n"                                           ,  deviceProp.pciDeviceID                     );
    ierr = PetscPrintf(PETSC_COMM_SELF," PCI domain ID of the device %d\n"                                           ,  deviceProp.pciDomainID                     );
    ierr = PetscPrintf(PETSC_COMM_SELF," 1 if device is a Tesla device using TCC driver, 0 otherwise %d\n"           ,  deviceProp.tccDriver                       );
    ierr = PetscPrintf(PETSC_COMM_SELF," Number of asynchronous engines %d\n"                                        ,  deviceProp.asyncEngineCount                );
    ierr = PetscPrintf(PETSC_COMM_SELF," Device shares a unified address space with the host %d\n"                   ,  deviceProp.unifiedAddressing               );
    ierr = PetscPrintf(PETSC_COMM_SELF," Peak memory clock frequency in kilohertz %d\n"                              ,  deviceProp.memoryClockRate                 );
    ierr = PetscPrintf(PETSC_COMM_SELF," Global memory bus width in bits %d\n"                                       ,  deviceProp.memoryBusWidth                  );
    ierr = PetscPrintf(PETSC_COMM_SELF," Size of L2 cache in bytes %d\n"                                             ,  deviceProp.l2CacheSize                     );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum resident threads per multiprocessor %d\n"                           ,  deviceProp.maxThreadsPerMultiProcessor     );
  } else {
    ierr = PetscPrintf(PETSC_COMM_SELF, " Unable to determine device %d properties, exiting\n",deviceNum);CHKERRQ(ierr);
    ierr = PetscPrintf(PETSC_COMM_SELF, " cuda Error: %s , exiting\n",hipGetErrorString( ierrCuda));CHKERRQ(ierr);
    return -1;
  }

  PetscLogEventRegister("ComputeFunction",0,&LogFunction); 
  ierr = PetscOptionsGetString(PETSC_NULL,"-da_vec_type",typeName,256,&flg);CHKERRQ(ierr);
  if (flg) {
    ierr = PetscStrstr(typeName,"cusp",&tmp);CHKERRQ(ierr);
    if (tmp) useCUSP = PETSC_TRUE;
  }

  size_t sizeIndex = 3 * sizeof(PetscInt);
  CUDA_SAFE_CALL(hipMalloc((void **) &cudaTest, sizeIndex));   // Allocate array on device

  //ierr = DMDACreate1d(PETSC_COMM_WORLD,DMDA_BOUNDARY_NONE,-8,1,1,PETSC_NULL,&da);CHKERRQ(ierr);
  PetscInt globalSize = 125;
  globalSize = 99;
  DM             da;
  ierr = DMDACreate3d(PETSC_COMM_WORLD,DMDA_BOUNDARY_NONE,DMDA_BOUNDARY_NONE,DMDA_BOUNDARY_NONE,DMDA_STENCIL_STAR,-globalSize,-globalSize,-globalSize,PETSC_DECIDE,PETSC_DECIDE,PETSC_DECIDE,1,1,PETSC_NULL,PETSC_NULL,PETSC_NULL,&da);CHKERRQ(ierr);
  ierr = DMCreateGlobalVector(da,&x); VecDuplicate(x,&f);CHKERRQ(ierr);
  if (useCUSP)
    {
     ierr = DMCreateMatrix(da,MATAIJCUSP,&J);CHKERRQ(ierr);
    }
  else
    {
     ierr = DMCreateMatrix(da,MATAIJ,&J);CHKERRQ(ierr);
    }

  PetscInt       GlobalDAMx,GlobalDAMy,GlobalDAMz,xs,xm,ys,ym,zs,zm;
  PetscScalar    hx,hy,hz;
  ierr = DMDAGetInfo(da,PETSC_IGNORE,&GlobalDAMx,&GlobalDAMy,&GlobalDAMz,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE);CHKERRQ(ierr);
  hx     = 1.0/(PetscReal)(GlobalDAMx-1);
  hy     = 1.0/(PetscReal)(GlobalDAMy-1);
  hz     = 1.0/(PetscReal)(GlobalDAMz-1);
  ierr = DMDAGetCorners(da,&xs,&ys,&zs,&xm,&ym,&zm);CHKERRQ(ierr);
  StarStencil  stencil_op(0,0,xs,ys,zs,xm,ym,zm,hx,hy,hz);// transformation operator
  ierr = DMSetApplicationContext(da,&stencil_op);CHKERRQ(ierr);

  ierr = SNESCreate(PETSC_COMM_WORLD,&snes);CHKERRQ(ierr);
  ierr = SNESSetFunction(snes,f,ComputeFunction,da);CHKERRQ(ierr);
  ierr = SNESSetJacobian(snes,J,J,ComputeJacobian,da);CHKERRQ(ierr);
  ierr = SNESSetFromOptions(snes);CHKERRQ(ierr);
  ierr = ComputeFunction(snes,x,f,(void *)da);
  ierr = SNESSolve(snes,PETSC_NULL,x);CHKERRQ(ierr);

  ierr = MatDestroy(&J);CHKERRQ(ierr);
  ierr = VecDestroy(&x);CHKERRQ(ierr);
  ierr = VecDestroy(&f);CHKERRQ(ierr);
  ierr = SNESDestroy(&snes);CHKERRQ(ierr);
  ierr = DMDestroy(&da);CHKERRQ(ierr);

  // call device reset to flush buffer
  CUDA_SAFE_CALL(hipDeviceReset());
  PetscFinalize();
  return 0;
}


PetscErrorCode ComputeFunction(SNES snes,Vec u,Vec f,void *ctx) 
{
  PetscInt       i,j,k;
  PetscInt       ustartshift,uendshift,xoffset,yoffset,zoffset,fstart;
  PetscScalar    ***uu,***ff,hxhzdhy,hyhzdhx,hxhydhz;
  PetscScalar    u_val,u_east,u_west,u_north,u_south,u_up, u_down, u_xx, u_yy,u_zz,sc ,two =2.0;
  DM             da = (DM) ctx; 
  Vec            ulocal;
  PetscErrorCode ierr;
  PetscMPIInt    rank,size;
  MPI_Comm       comm;
  CUSPARRAY      *uarray,*farray;
  PetscLogEventBegin(LogFunction,0,0,0,0); // init libMesh

  ierr = DMGetLocalVector(da,&ulocal);CHKERRQ(ierr);
  ierr = DMGlobalToLocalBegin(da,u,INSERT_VALUES,ulocal);CHKERRQ(ierr);
  ierr = DMGlobalToLocalEnd(da,u,INSERT_VALUES,ulocal);CHKERRQ(ierr);
  StarStencil  *stencil_op;
  ierr = DMGetApplicationContext(da,(void *)&stencil_op);CHKERRQ(ierr);
  hxhzdhy = stencil_op->m_hx*stencil_op->m_hz/stencil_op->m_hy;
  hyhzdhx = stencil_op->m_hy*stencil_op->m_hz/stencil_op->m_hx;
  hxhydhz = stencil_op->m_hx*stencil_op->m_hy/stencil_op->m_hz;
  sc      = stencil_op->m_hx*stencil_op->m_hy*stencil_op->m_hz*3.0;

  if (useCUSP) {
    ierr = VecCUSPGetArrayRead(ulocal,&uarray);CHKERRQ(ierr);
    ierr = VecCUSPGetArrayWrite(f,&farray);CHKERRQ(ierr);
    ierr = PetscObjectGetComm((PetscObject)da,&comm);CHKERRQ(ierr);
    ierr = MPI_Comm_size(comm,&size);CHKERRQ(ierr);
    ierr = MPI_Comm_rank(comm,&rank);CHKERRQ(ierr);
    if (rank) ustartshift = 1; else ustartshift = 0;
    if (rank != size-1) uendshift = 1; else uendshift = 0;
    xoffset = 1;
    yoffset = stencil_op->m_xm;
    zoffset = stencil_op->m_xm*stencil_op->m_ym;
    ierr = VecGetOwnershipRange(f,&fstart,PETSC_NULL);CHKERRQ(ierr);
    try {
      
      // typedef these iterators for shorthand
      thrust::for_each(
		       thrust::make_zip_iterator(
						 thrust::make_tuple(
            farray->begin(),                              //0
            uarray->begin()+ustartshift,                  //1  u(i  ,j  ,k  )
            uarray->begin()+ustartshift + xoffset,        //2  u(i+1,j  ,k  )
            uarray->begin()+ustartshift - xoffset,        //3  u(i-1,j  ,k  )
            uarray->begin()+ustartshift + yoffset,        //4  u(i  ,j+1,k  )
            uarray->begin()+ustartshift - yoffset,        //5  u(i  ,j-1,k  )
            uarray->begin()+ustartshift + zoffset,        //6  u(i  ,j  ,k+1)
            uarray->begin()+ustartshift - zoffset,        //7  u(i  ,j  ,k-1)
            thrust::counting_iterator<int>(fstart)        //8
                                                                    )), 
		       thrust::make_zip_iterator(
						 thrust::make_tuple(
            farray->end(),                            //0
            //farray->begin()+10,                            //0
            uarray->end()+uendshift,                  //1  u(i  ,j  ,k  )
            uarray->end()+uendshift + xoffset,        //2  u(i+1,j  ,k  )
            uarray->end()+uendshift - xoffset,        //3  u(i-1,j  ,k  )
            uarray->end()+uendshift + yoffset,        //4  u(i  ,j+1,k  )
            uarray->end()+uendshift - yoffset,        //5  u(i  ,j-1,k  )
            uarray->end()+uendshift + zoffset,        //6  u(i  ,j  ,k+1)
            uarray->end()+uendshift - zoffset,        //7  u(i  ,j  ,k-1)
            thrust::counting_iterator<int>(fstart) + u->map->n        //8
                                                                    )),
		       *stencil_op);
      
      PetscInt hostTest[3]={-1,-1,-1};
      //CUDA_SAFE_CALL(hipMemcpy(hostTest, cudaTest,3*sizeof(PetscInt),hipMemcpyDeviceToHost));
      ierr = PetscPrintf(PETSC_COMM_WORLD, "%d %d %d \n",hostTest[0],hostTest[1],hostTest[2]);CHKERRQ(ierr);
    }
    catch(char* all){
      ierr = PetscPrintf(PETSC_COMM_WORLD, "Thrust is not working\n");CHKERRQ(ierr);
    }
    ierr = VecCUSPRestoreArrayRead(ulocal,&uarray);CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayWrite(f,&farray);CHKERRQ(ierr);
  } else {
    ierr = DMDAVecGetArray(da,ulocal,&uu);CHKERRQ(ierr);
    ierr = DMDAVecGetArray(da,f,&ff);CHKERRQ(ierr);
    
    PetscInt       GlobalDAMx,GlobalDAMy,GlobalDAMz;
    ierr = DMDAGetInfo(da,PETSC_IGNORE,&GlobalDAMx,&GlobalDAMy,&GlobalDAMz,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE);CHKERRQ(ierr);
    /* Compute function over the locally owned part of the grid */
    for (k=stencil_op->m_zs; k<stencil_op->m_zs+stencil_op->m_zm; k++) {
      for (j=stencil_op->m_ys; j<stencil_op->m_ys+stencil_op->m_ym; j++) {
        for (i=stencil_op->m_xs; i<stencil_op->m_xs+stencil_op->m_xm; i++) {
          if (i == 0 || j == 0 || k == 0 || i == GlobalDAMx-1 || j == GlobalDAMy-1 || k == GlobalDAMz-1) {
            ff[k][j][i] = uu[k][j][i];
          } else {
            u_val       = uu[k][j][i];
            u_east      = uu[k][j][i+1];
            u_west      = uu[k][j][i-1];
            u_north     = uu[k][j+1][i];
            u_south     = uu[k][j-1][i];
            u_up        = uu[k+1][j][i];
            u_down      = uu[k-1][j][i];
            u_xx        = (-u_east  + two*u_val - u_west )*hyhzdhx;
            u_yy        = (-u_north + two*u_val - u_south)*hxhzdhy;
            u_zz        = (-u_up    + two*u_val - u_down )*hxhydhz;
            ff[k][j][i]  = u_xx + u_yy + u_zz - sc*PetscExpScalar(u_val);
          }
        }
      }
    }
    ierr = DMDAVecRestoreArray(da,ulocal,&uu);CHKERRQ(ierr);
    ierr = DMDAVecRestoreArray(da,f,&ff);CHKERRQ(ierr);
  }
  ierr = DMRestoreLocalVector(da,&ulocal);CHKERRQ(ierr);
  PetscLogEventEnd(LogFunction,0,0,0,0);   // init libMesh
  //VecView(u,0);printf("f\n");
  //VecView(f,0);
  return 0;

}
PetscErrorCode ComputeJacobian(SNES snes,Vec x,Mat *J,Mat *B,MatStructure *flag,void *ctx)
{
  DM             da = (DM) ctx; 
  Vec            xlocal;
  PetscErrorCode ierr;

  ierr = DMGetLocalVector(da,&xlocal);DMGlobalToLocalBegin(da,x,INSERT_VALUES,xlocal);CHKERRQ(ierr);
  ierr = DMGlobalToLocalEnd(da,x,INSERT_VALUES,xlocal);CHKERRQ(ierr);

  PetscInt       GlobalDAMx,GlobalDAMy,GlobalDAMz,xs,xm,ys,ym,zs,zm;
  PetscScalar    hx,hy,hz;
  ierr = DMDAGetInfo(da,PETSC_IGNORE,&GlobalDAMx,&GlobalDAMy,&GlobalDAMz,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE);CHKERRQ(ierr);
  hx     = 1.0/(PetscReal)(GlobalDAMx-1);
  hy     = 1.0/(PetscReal)(GlobalDAMy-1);
  hz     = 1.0/(PetscReal)(GlobalDAMz-1);
  ierr = DMDAGetCorners(da,&xs,&ys,&zs,&xm,&ym,&zm);CHKERRQ(ierr);

  PetscScalar    hxhzdhy,hyhzdhx,hxhydhz,sc;
  hxhzdhy = hx*hz/hy;
  hyhzdhx = hy*hz/hx;
  hxhydhz = hx*hy/hz;
  sc      = hx*hy*hz*3.0;

  ierr = MatZeroEntries(*J);CHKERRQ(ierr);
  ierr = MatShift(*J,1.0);CHKERRQ(ierr);

  StarStencil  *stencil_op;
  ierr = DMGetApplicationContext(da,(void *)&stencil_op);CHKERRQ(ierr);

  /* Compute function over the locally owned part of the grid */
  PetscScalar    v[7],two = 2.0;
  MatStencil     col[7],row;
  PetscInt       i,j,k;
  for (k=zs; k<zs+zm; k++) {
    for (j=ys; j<ys+ym; j++) {
      for (i=xs; i<xs+xm; i++) {
        row.k = k; row.j = j; row.i = i;
        if (i > 0 && j > 0 && k > 0 && i < GlobalDAMx-1 && j < GlobalDAMy-1 && k < GlobalDAMz-1) {
          v[0] = -0.5 * stencil_op->m_conduction * hxhydhz; col[0].k=k-1;col[0].j=j;  col[0].i = i;
          v[1] = -0.5 * stencil_op->m_conduction * hxhzdhy; col[1].k=k;  col[1].j=j-1;col[1].i = i;
          v[2] = -0.5 * stencil_op->m_conduction * hyhzdhx; col[2].k=k;  col[2].j=j;  col[2].i = i-1;
          v[3] =  sc*(  stencil_op->m_density*stencil_op->m_specificheat/stencil_op->m_deltat 
                    + 0.5 * stencil_op->m_perfusion * stencil_op->m_bloodspecificheat) 
                    + 1.0 * stencil_op->m_bloodspecificheat * (hyhzdhx+hxhzdhy+hxhydhz);
                           col[3].k=row.k;col[3].j=row.j;col[3].i = row.i;
          v[4] = -0.5 * stencil_op->m_conduction * hyhzdhx; col[4].k=k;  col[4].j=j;  col[4].i = i+1;
          v[5] = -0.5 * stencil_op->m_conduction * hxhzdhy; col[5].k=k;  col[5].j=j+1;col[5].i = i;
          v[6] = -0.5 * stencil_op->m_conduction * hxhydhz; col[6].k=k+1;col[6].j=j;  col[6].i = i;
          ierr = MatSetValuesStencil(*J,1,&row,7,col,v,INSERT_VALUES);CHKERRQ(ierr);
        }
      }
    }
  }

  ierr = MatAssemblyBegin(*J,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);
  ierr = MatAssemblyEnd(*J,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);
  *flag = SAME_NONZERO_PATTERN;
  ierr = DMRestoreLocalVector(da,&xlocal);CHKERRQ(ierr);
  return 0;}

