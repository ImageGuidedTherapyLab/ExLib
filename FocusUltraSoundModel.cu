#include "hip/hip_runtime.h"
static char help[] = "Solves -Laplacian u - exp(u) = 0,  0 < x < 1 using GPU\n\n";
/*
   Same as ex47.c except it also uses the GPU to evaluate the function
*/

#include <petscdmda.h>
#include <petscsnes.h>
#include <petsccusp.h>
#include "cusp/detail/device/utils.h"

extern PetscErrorCode ComputeFunction(SNES,Vec,Vec,void*), ComputeJacobian(SNES,Vec,Mat*,Mat*,MatStructure*,void*);
PetscBool  useCUSP = PETSC_FALSE;
PetscLogEvent LogFunction = 0;
__device__ PetscInt *cudaTest;

int main(int argc,char **argv) 
{
  SNES           snes; 
  Vec            x,f;  
  Mat            J;
  DM             da;
  PetscErrorCode ierr;
  hipError_t      ierrCuda;
  char           *tmp,typeName[256];
  int            myrank;
  PetscBool      flg;

  PetscInitialize(&argc,&argv,(char *)0,help);

  MPI_Comm_rank(PETSC_COMM_WORLD, &myrank);
  int deviceNum=myrank;
  {
    int deviceCount;
    CUDA_SAFE_CALL(hipGetDeviceCount(&deviceCount));
    
    ierr = PetscPrintf(PETSC_COMM_SELF, "!!!!!found %d devices !!!!!\n",deviceCount);CHKERRQ(ierr);
    if (deviceCount == 0) {
      ierr = PetscPrintf(PETSC_COMM_SELF, "!!!!!No devices found!!!!!\n");CHKERRQ(ierr);
      return -1000;
    }

    if (deviceNum >= deviceCount || deviceNum < 0) {
      ierr = PetscPrintf(PETSC_COMM_SELF, "\n!!!!!Invalid GPU number %d given hence default gpu %d will be used !!!!!\n", deviceNum, 0);CHKERRQ(ierr);
      deviceNum = 0;
    }
  }

  ierrCuda =  hipSetDevice(deviceNum);
  if (ierrCuda != hipSuccess) {
    ierr = PetscPrintf(PETSC_COMM_SELF, " cuda Error: %s , exiting\n",hipGetErrorString( ierrCuda));CHKERRQ(ierr);
    return -1;
  }
  ierr = PetscPrintf(PETSC_COMM_SELF, " reseting GPU: \n");CHKERRQ(ierr);
  CUDA_SAFE_CALL(hipDeviceReset());

  ierr = PetscPrintf(PETSC_COMM_SELF, "Running on...\n\n");CHKERRQ(ierr);
  hipDeviceProp_t deviceProp;
  if (hipGetDeviceProperties(&deviceProp, deviceNum) == hipSuccess) {
    ierr = PetscPrintf(PETSC_COMM_SELF, " Device %d: %s %d.%d\n", deviceNum, deviceProp.name,deviceProp.major,deviceProp.minor);CHKERRQ(ierr);
    ierr = PetscPrintf(PETSC_COMM_SELF," Global memory available on device in bytes %d\n"                            ,  deviceProp.totalGlobalMem                  );
    ierr = PetscPrintf(PETSC_COMM_SELF," Shared memory available per block in bytes %d\n"                            ,  deviceProp.sharedMemPerBlock               );
    ierr = PetscPrintf(PETSC_COMM_SELF," 32-bit registers available per block %d\n"                                  ,  deviceProp.regsPerBlock                    );
    ierr = PetscPrintf(PETSC_COMM_SELF," Warp size in threads %d\n"                                                  ,  deviceProp.warpSize                        );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum pitch in bytes allowed by memory copies %d\n"                       ,  deviceProp.memPitch                        );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum number of threads per block %d\n"                                   ,  deviceProp.maxThreadsPerBlock              );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum size of each dimension of a block %d\n"                             ,  deviceProp.maxThreadsDim[0]                );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum size of each dimension of a block %d\n"                             ,  deviceProp.maxThreadsDim[1]                );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum size of each dimension of a block %d\n"                             ,  deviceProp.maxThreadsDim[2]                );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum size of each dimension of a grid %d\n"                              ,  deviceProp.maxGridSize[0]                  );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum size of each dimension of a grid %d\n"                              ,  deviceProp.maxGridSize[1]                  );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum size of each dimension of a grid %d\n"                              ,  deviceProp.maxGridSize[2]                  );
    ierr = PetscPrintf(PETSC_COMM_SELF," Clock frequency in kilohertz %d\n"                                          ,  deviceProp.clockRate                       );
    ierr = PetscPrintf(PETSC_COMM_SELF," Constant memory available on device in bytes %d\n"                          ,  deviceProp.totalConstMem                   );
    ierr = PetscPrintf(PETSC_COMM_SELF," Alignment requirement for textures %d\n"                                    ,  deviceProp.textureAlignment                );
    ierr = PetscPrintf(PETSC_COMM_SELF," Number of multiprocessors on device %d\n"                                   ,  deviceProp.multiProcessorCount             );
    ierr = PetscPrintf(PETSC_COMM_SELF," Specified whether there is a run time limit on kernels %d\n"                ,  deviceProp.kernelExecTimeoutEnabled        );
    ierr = PetscPrintf(PETSC_COMM_SELF," Device is integrated as opposed to discrete %d\n"                           ,  deviceProp.integrated                      );
    ierr = PetscPrintf(PETSC_COMM_SELF," Device can map host memory with hipHostAlloc/hipHostGetDevicePointer %d\n",  deviceProp.canMapHostMemory                );
    ierr = PetscPrintf(PETSC_COMM_SELF," Compute mode (See ::hipComputeMode) %d\n"                                  ,  deviceProp.computeMode                     );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 1D texture size %d\n"                                               ,  deviceProp.maxTexture1D                    );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 2D texture dimensions %d\n"                                         ,  deviceProp.maxTexture2D[0]                 );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 2D texture dimensions %d\n"                                         ,  deviceProp.maxTexture2D[1]                 );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 3D texture dimensions %d\n"                                         ,  deviceProp.maxTexture3D[0]                 );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 3D texture dimensions %d\n"                                         ,  deviceProp.maxTexture3D[1]                 );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 3D texture dimensions %d\n"                                         ,  deviceProp.maxTexture3D[2]                 );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 1D layered texture dimensions %d\n"                                 ,  deviceProp.maxTexture1DLayered[0]          );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 1D layered texture dimensions %d\n"                                 ,  deviceProp.maxTexture1DLayered[1]          );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 2D layered texture dimensions %d\n"                                 ,  deviceProp.maxTexture2DLayered[0]          );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 2D layered texture dimensions %d\n"                                 ,  deviceProp.maxTexture2DLayered[1]          );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 2D layered texture dimensions %d\n"                                 ,  deviceProp.maxTexture2DLayered[2]          );
    ierr = PetscPrintf(PETSC_COMM_SELF," Alignment requirements for surfaces %d\n"                                   ,  deviceProp.surfaceAlignment                );
    ierr = PetscPrintf(PETSC_COMM_SELF," Device can possibly execute multiple kernels concurrently %d\n"             ,  deviceProp.concurrentKernels               );
    ierr = PetscPrintf(PETSC_COMM_SELF," Device has ECC support enabled %d\n"                                        ,  deviceProp.ECCEnabled                      );
    ierr = PetscPrintf(PETSC_COMM_SELF," PCI bus ID of the device %d\n"                                              ,  deviceProp.pciBusID                        );
    ierr = PetscPrintf(PETSC_COMM_SELF," PCI device ID of the device %d\n"                                           ,  deviceProp.pciDeviceID                     );
    ierr = PetscPrintf(PETSC_COMM_SELF," PCI domain ID of the device %d\n"                                           ,  deviceProp.pciDomainID                     );
    ierr = PetscPrintf(PETSC_COMM_SELF," 1 if device is a Tesla device using TCC driver, 0 otherwise %d\n"           ,  deviceProp.tccDriver                       );
    ierr = PetscPrintf(PETSC_COMM_SELF," Number of asynchronous engines %d\n"                                        ,  deviceProp.asyncEngineCount                );
    ierr = PetscPrintf(PETSC_COMM_SELF," Device shares a unified address space with the host %d\n"                   ,  deviceProp.unifiedAddressing               );
    ierr = PetscPrintf(PETSC_COMM_SELF," Peak memory clock frequency in kilohertz %d\n"                              ,  deviceProp.memoryClockRate                 );
    ierr = PetscPrintf(PETSC_COMM_SELF," Global memory bus width in bits %d\n"                                       ,  deviceProp.memoryBusWidth                  );
    ierr = PetscPrintf(PETSC_COMM_SELF," Size of L2 cache in bytes %d\n"                                             ,  deviceProp.l2CacheSize                     );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum resident threads per multiprocessor %d\n"                           ,  deviceProp.maxThreadsPerMultiProcessor     );
  } else {
    ierr = PetscPrintf(PETSC_COMM_SELF, " Unable to determine device %d properties, exiting\n",deviceNum);CHKERRQ(ierr);
    ierr = PetscPrintf(PETSC_COMM_SELF, " cuda Error: %s , exiting\n",hipGetErrorString( ierrCuda));CHKERRQ(ierr);
    return -1;
  }

  PetscLogEventRegister("ComputeFunction",0,&LogFunction); 
  ierr = PetscOptionsGetString(PETSC_NULL,"-da_vec_type",typeName,256,&flg);CHKERRQ(ierr);
  if (flg) {
    ierr = PetscStrstr(typeName,"cusp",&tmp);CHKERRQ(ierr);
    if (tmp) useCUSP = PETSC_TRUE;
  }

  size_t sizeIndex = 3 * sizeof(PetscInt);
  CUDA_SAFE_CALL(hipMalloc((void **) &cudaTest, sizeIndex));   // Allocate array on device

  //ierr = DMDACreate1d(PETSC_COMM_WORLD,DMDA_BOUNDARY_NONE,-8,1,1,PETSC_NULL,&da);CHKERRQ(ierr);
  PetscInt globalSize = 4;
  ierr = DMDACreate3d(PETSC_COMM_WORLD,DMDA_BOUNDARY_NONE,DMDA_BOUNDARY_NONE,DMDA_BOUNDARY_NONE,DMDA_STENCIL_STAR,-globalSize,-globalSize,-globalSize,PETSC_DECIDE,PETSC_DECIDE,PETSC_DECIDE,1,1,PETSC_NULL,PETSC_NULL,PETSC_NULL,&da);CHKERRQ(ierr);
  ierr = DMCreateGlobalVector(da,&x); VecDuplicate(x,&f);CHKERRQ(ierr);
  ierr = DMCreateMatrix(da,MATAIJ,&J);CHKERRQ(ierr);

  ierr = SNESCreate(PETSC_COMM_WORLD,&snes);CHKERRQ(ierr);
  ierr = SNESSetFunction(snes,f,ComputeFunction,da);CHKERRQ(ierr);
  ierr = SNESSetJacobian(snes,J,J,ComputeJacobian,da);CHKERRQ(ierr);
  ierr = SNESSetFromOptions(snes);CHKERRQ(ierr);
  ierr = ComputeFunction(snes,x,f,(void *)da);
  //ierr = SNESSolve(snes,PETSC_NULL,x);CHKERRQ(ierr);

  ierr = MatDestroy(&J);CHKERRQ(ierr);
  ierr = VecDestroy(&x);CHKERRQ(ierr);
  ierr = VecDestroy(&f);CHKERRQ(ierr);
  ierr = SNESDestroy(&snes);CHKERRQ(ierr);
  ierr = DMDestroy(&da);CHKERRQ(ierr);

  PetscFinalize();
  return 0;
}

struct StarStencil
{
  PetscInt       m_rank,m_deviceNum; //device info
  PetscInt       m_xs,m_ys,m_zs,m_xm,m_ym,m_zm; //corners
  PetscScalar    m_hxhzdhy,m_hyhzdhx,m_hxhydhz;
  
  StarStencil(PetscInt rank, PetscInt deviceNum,
              PetscInt  xs,PetscInt  ys,PetscInt  zs, 
              PetscInt  xm,PetscInt  ym,PetscInt  zm,
              PetscScalar hxhzdhy,PetscScalar hyhzdhx,PetscScalar hxhydhz) : 
               m_rank(rank),m_deviceNum(deviceNum),
               m_xs(xs),m_ys(ys),m_zs(zs), 
               m_xm(xm),m_ym(ym),m_zm(zm), 
               m_hxhzdhy(m_hxhzdhy),m_hyhzdhx(hyhzdhx),m_hxhydhz(hxhydhz) {}

	template <typename Tuple>
	__host__ __device__
	void operator()(Tuple t)
	{
		/* f = (2*u_i - u_(i+1) - u_(i-1))/h - h*exp(u_i) */
	     thrust::get<0>(t) = 1;
             PetscInt Iz = thrust::get<8>(t)/m_ym/m_xm;
             PetscInt Iy = (thrust::get<8>(t)-Iz*m_ym*m_xm)/m_xm;
             PetscInt Ix = (thrust::get<8>(t)-Iz*m_ym*m_xm- Iy*m_xm);
             // print launch parameters and dbg info
             printf("rank=%d device=%d blockDim=(%d,%d,%d) gridDim=(%d,%d,%d) warpSize=%d blockIdx=(%d,%d,%d) threadIdx=(%d,%d,%d) size=(%d,%d,%d) globalID=%d index=(%d,%d,%d)\n",m_rank,m_deviceNum,blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z, warpSize,blockIdx.x,blockIdx.y,blockIdx.z,threadIdx.x,threadIdx.y,threadIdx.z,m_xm,m_ym,m_zm,thrust::get<8>(t),Ix,Iy,Iz);
             if (Ix > 0  && Ix < m_xm-1) {
               thrust::get<0>(t) = (2.0*thrust::get<1>(t) - thrust::get<2>(t) - thrust::get<3>(t)) / m_hxhzdhy - m_hxhydhz*exp(thrust::get<1>(t));
             } else if (Ix == 0) {
               thrust::get<0>(t) = thrust::get<1>(t) / m_hxhzdhy;
             } else if (Ix == m_xm-1) {
               thrust::get<0>(t) = thrust::get<1>(t) / m_hxhzdhy;
             } 
		
	}
};

PetscErrorCode ComputeFunction(SNES snes,Vec u,Vec f,void *ctx) 
{
  PetscInt       i,j,k,GlobalDAMx,GlobalDAMy,GlobalDAMz,xs,xm,ys,ym,zs,zm;
  PetscInt       ustartshift,uendshift,xoffset,yoffset,zoffset,fstart;
  PetscScalar    ***uu,***ff,hx,hy,hz, hxhzdhy,hyhzdhx,hxhydhz;
  PetscScalar    u_val,u_east,u_west,u_north,u_south,u_up, u_down, u_xx, u_yy,u_zz,sc ,two =2.0;
  DM             da = (DM) ctx; 
  Vec            ulocal;
  PetscErrorCode ierr;
  PetscMPIInt    rank,size;
  MPI_Comm       comm;
  CUSPARRAY      *uarray,*farray;
  PetscLogEventBegin(LogFunction,0,0,0,0); // init libMesh

  ierr = DMDAGetInfo(da,PETSC_IGNORE,&GlobalDAMx,&GlobalDAMy,&GlobalDAMz,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE);CHKERRQ(ierr);
  hx     = 1.0/(PetscReal)(GlobalDAMx-1);
  hy     = 1.0/(PetscReal)(GlobalDAMy-1);
  hz     = 1.0/(PetscReal)(GlobalDAMz-1);
  hxhzdhy = hx*hz/hy;
  hyhzdhx = hy*hz/hx;
  hxhydhz = hx*hy/hz;
  sc     = hx*hy*hz*3.0;
  ierr = DMGetLocalVector(da,&ulocal);CHKERRQ(ierr);
  ierr = DMGlobalToLocalBegin(da,u,INSERT_VALUES,ulocal);CHKERRQ(ierr);
  ierr = DMGlobalToLocalEnd(da,u,INSERT_VALUES,ulocal);CHKERRQ(ierr);

  ierr = DMDAGetCorners(da,&xs,&ys,&zs,&xm,&ym,&zm);CHKERRQ(ierr);
  if (useCUSP) {
    StarStencil  stencil_op(0,0,xs,ys,zs,xm,ym,zm,hxhzdhy,hyhzdhx,hxhydhz);// transformation operator
    ierr = VecCUSPGetArrayRead(ulocal,&uarray);CHKERRQ(ierr);
    ierr = VecCUSPGetArrayWrite(f,&farray);CHKERRQ(ierr);
    ierr = PetscObjectGetComm((PetscObject)da,&comm);CHKERRQ(ierr);
    ierr = MPI_Comm_size(comm,&size);CHKERRQ(ierr);
    ierr = MPI_Comm_rank(comm,&rank);CHKERRQ(ierr);
    if (rank) ustartshift = 1; else ustartshift = 0;
    if (rank != size-1) uendshift = 1; else uendshift = 0;
    xoffset = 1;
    yoffset = xm;
    zoffset = xm*ym;
    ierr = VecGetOwnershipRange(f,&fstart,PETSC_NULL);CHKERRQ(ierr);
    try {
      
      // typedef these iterators for shorthand
      thrust::for_each(
		       thrust::make_zip_iterator(
						 thrust::make_tuple(
            farray->begin(),                              //0
            uarray->begin()+ustartshift,                  //1  u(i  ,j  ,k  )
            uarray->begin()+ustartshift + xoffset,        //2  u(i+1,j  ,k  )
            uarray->begin()+ustartshift - xoffset,        //3  u(i-1,j  ,k  )
            uarray->begin()+ustartshift + yoffset,        //4  u(i  ,j+1,k  )
            uarray->begin()+ustartshift - yoffset,        //5  u(i  ,j-1,k  )
            uarray->begin()+ustartshift + zoffset,        //6  u(i  ,j  ,k+1)
            uarray->begin()+ustartshift - zoffset,        //7  u(i  ,j  ,k-1)
            thrust::counting_iterator<int>(fstart)        //8
                                                                    )), 
		       thrust::make_zip_iterator(
						 thrust::make_tuple(
            farray->end(),                            //0
            //farray->begin()+10,                            //0
            uarray->end()+uendshift,                  //1  u(i  ,j  ,k  )
            uarray->end()+uendshift + xoffset,        //2  u(i+1,j  ,k  )
            uarray->end()+uendshift - xoffset,        //3  u(i-1,j  ,k  )
            uarray->end()+uendshift + yoffset,        //4  u(i  ,j+1,k  )
            uarray->end()+uendshift - yoffset,        //5  u(i  ,j-1,k  )
            uarray->end()+uendshift + zoffset,        //6  u(i  ,j  ,k+1)
            uarray->end()+uendshift - zoffset,        //7  u(i  ,j  ,k-1)
            thrust::counting_iterator<int>(fstart) + u->map->n        //8
                                                                    )),
		       stencil_op);
      
      PetscInt hostTest[3]={-1,-1,-1};
      //CUDA_SAFE_CALL(hipMemcpy(hostTest, cudaTest,3*sizeof(PetscInt),hipMemcpyDeviceToHost));
      ierr = PetscPrintf(PETSC_COMM_WORLD, "%d %d %d \n",hostTest[0],hostTest[1],hostTest[2]);CHKERRQ(ierr);
    }
    catch(char* all){
      ierr = PetscPrintf(PETSC_COMM_WORLD, "Thrust is not working\n");CHKERRQ(ierr);
    }
    ierr = VecCUSPRestoreArrayRead(ulocal,&uarray);CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayWrite(f,&farray);CHKERRQ(ierr);
  } else {
    ierr = DMDAVecGetArray(da,ulocal,&uu);CHKERRQ(ierr);
    ierr = DMDAVecGetArray(da,f,&ff);CHKERRQ(ierr);
    
    /* Compute function over the locally owned part of the grid */
    for (k=zs; k<zs+zm; k++) {
      for (j=ys; j<ys+ym; j++) {
        for (i=xs; i<xs+xm; i++) {
          if (i == 0 || j == 0 || k == 0 || i == GlobalDAMx-1 || j == GlobalDAMy-1 || k == GlobalDAMz-1) {
            ff[k][j][i] = uu[k][j][i];
          } else {
            u_val       = uu[k][j][i];
            u_east      = uu[k][j][i+1];
            u_west      = uu[k][j][i-1];
            u_north     = uu[k][j+1][i];
            u_south     = uu[k][j-1][i];
            u_up        = uu[k+1][j][i];
            u_down      = uu[k-1][j][i];
            u_xx        = (-u_east  + two*u_val - u_west )*hyhzdhx;
            u_yy        = (-u_north + two*u_val - u_south)*hxhzdhy;
            u_zz        = (-u_up    + two*u_val - u_down )*hxhydhz;
            ff[k][j][i]  = u_xx + u_yy + u_zz - sc*PetscExpScalar(u_val);
          }
        }
      }
    }
    ierr = DMDAVecRestoreArray(da,ulocal,&uu);CHKERRQ(ierr);
    ierr = DMDAVecRestoreArray(da,f,&ff);CHKERRQ(ierr);
  }
  ierr = DMRestoreLocalVector(da,&ulocal);CHKERRQ(ierr);
  PetscLogEventEnd(LogFunction,0,0,0,0);   // init libMesh
  //VecView(u,0);printf("f\n");
  //VecView(f,0);
  return 0;

}
PetscErrorCode ComputeJacobian(SNES snes,Vec x,Mat *J,Mat *B,MatStructure *flag,void *ctx)
{
  DM             da = (DM) ctx; 
  PetscInt       i,Mx,xm,xs; 
  PetscScalar    hx,*xx; 
  Vec            xlocal;
  PetscErrorCode ierr;

  ierr = DMDAGetInfo(da,PETSC_IGNORE,&Mx,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE);CHKERRQ(ierr);
  hx = 1.0/(PetscReal)(Mx-1);
  ierr = DMGetLocalVector(da,&xlocal);DMGlobalToLocalBegin(da,x,INSERT_VALUES,xlocal);CHKERRQ(ierr);
  ierr = DMGlobalToLocalEnd(da,x,INSERT_VALUES,xlocal);CHKERRQ(ierr);
  ierr = DMDAVecGetArray(da,xlocal,&xx);CHKERRQ(ierr);
  ierr = DMDAGetCorners(da,&xs,PETSC_NULL,PETSC_NULL,&xm,PETSC_NULL,PETSC_NULL);CHKERRQ(ierr);

  ierr = MatZeroEntries(*J);CHKERRQ(ierr);
  ierr = MatShift(*J,1.0);CHKERRQ(ierr);
  for (i=xs; i<xs+xm; i++) {
    if (i == 0 || i == Mx-1) { 
      ierr = MatSetValue(*J,i,i,1.0/hx,INSERT_VALUES);CHKERRQ(ierr);
    } else {
      ierr = MatSetValue(*J,i,i-1,-1.0/hx,INSERT_VALUES);CHKERRQ(ierr);
      ierr = MatSetValue(*J,i,i,2.0/hx - hx*PetscExpScalar(xx[i]),INSERT_VALUES);CHKERRQ(ierr);
      ierr = MatSetValue(*J,i,i+1,-1.0/hx,INSERT_VALUES);CHKERRQ(ierr);
    }
  }
  ierr = MatAssemblyBegin(*J,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);
  ierr = MatAssemblyEnd(*J,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);
  *flag = SAME_NONZERO_PATTERN;
  ierr = DMDAVecRestoreArray(da,xlocal,&xx);CHKERRQ(ierr);
  ierr = DMRestoreLocalVector(da,&xlocal);CHKERRQ(ierr);
  return 0;}

